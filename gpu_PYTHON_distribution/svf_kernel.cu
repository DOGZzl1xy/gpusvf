
#include <hip/hip_runtime.h>
// svf_kernel.cu
#define PI 3.1415926f

__device__ float annulus_weight(float altitude, float aziinterval) {
    float n = 90.0f;
    float steprad = (360.0f/aziinterval) * PI/180.0f;
    float annulus = 91.0f - altitude;
    float w = 1.0f/(2.0f*PI) * sinf(PI / (2.0f*n)) * sinf((PI * (2.0f * annulus - 1.0f)) / (2.0f * n));
    return steprad * w;
}

extern "C" __global__ void svf_shadowcasting_cupy(
    float* svf_Latt, float* dsm_Latt, float* tree_Latt, float scale, int imageW, int imageH, int rangeDist) // 新增 rangeDist 参数
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= imageW || y >= imageH) return;

    int index4 = x + y * imageW;

    // 如果当前位置有树且树高>0，认为站在树下，SVF=0 (性能优化，予以保留)
    if (tree_Latt[index4] > 0.0f) {
        svf_Latt[index4] = 0.03f;
        return;
    }

    // 穹顶分区参数
    float iangle[8] = {6, 18, 30, 42, 54, 66, 78, 90};
    float aziinterval[8] = {30, 30, 24, 24, 18, 12, 6, 1};
    float annulino[9] = {0, 12, 24, 36, 48, 60, 72, 84, 90};

    float svf = 0.0f;
    int idx = 0;
    // 计算每个分区的SVF
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < (int)aziinterval[i]; j++) {
            float altitude = iangle[i];
            float azimuth = j * (360.0f/aziinterval[i]);
            float altitude_rad = PI * altitude / 180.0f;
            float theta;
            // 太阳方位角转换
            if (azimuth < 90.0f && azimuth > 0.0f) {
                theta = PI * (90.0f - azimuth) / 180.0f;
            } else {
                theta = PI * (450.0f - azimuth) / 180.0f;
            }

            // --- 核心算法修改 ---
            float f = dsm_Latt[index4]; // 视线高度初始化为当前点高程
            float h_orig = f; // 保存原始地面高度

            // 修改1: 使用动态的 rangeDist，并从 radius=1.0f 开始避免采样自身
            for (float radius = 1.0f; radius < rangeDist; radius += 1.0f) {
                float x_f = x + radius * cosf(theta);
                float y_f = y - radius * sinf(theta);

                // 检查浮点坐标是否越界，并为双线性插值留出1个像素的边界
                if (x_f < 0.0f || x_f >= imageW - 1.0f || y_f < 0.0f || y_f >= imageH - 1.0f) break;

                // 修改2: 使用双线性插值法，解决边缘突变问题
                int x1 = (int)floorf(x_f);
                int y1 = (int)floorf(y_f);

                float x_frac = x_f - x1;
                float y_frac = y_f - y1;

                // 读取周围4个像素的高度和树高
                float h11 = dsm_Latt[x1 + y1 * imageW];
                float t11 = tree_Latt[x1 + y1 * imageW];
                if (t11 > 0.0f) h11 += 6.0f;

                float h12 = dsm_Latt[x1 + (y1 + 1) * imageW];
                float t12 = tree_Latt[x1 + (y1 + 1) * imageW];
                if (t12 > 0.0f) h12 += 6.0f;

                float h21 = dsm_Latt[(x1 + 1) + y1 * imageW];
                float t21 = tree_Latt[(x1 + 1) + y1 * imageW];
                if (t21 > 0.0f) h21 += 6.0f;

                float h22 = dsm_Latt[(x1 + 1) + (y1 + 1) * imageW];
                float t22 = tree_Latt[(x1 + 1) + (y1 + 1) * imageW];
                if (t22 > 0.0f) h22 += 6.0f;

                // 对高度进行双线性插值
                float h_top = h11 * (1.0f - x_frac) + h21 * x_frac;
                float h_bottom = h12 * (1.0f - x_frac) + h22 * x_frac;
                float height1 = h_top * (1.0f - y_frac) + h_bottom * y_frac;

                // 寻找路径上的最大遮挡高度角
                float temp = height1 - radius * tanf(altitude_rad) / scale;
                if (f < temp) f = temp;

                // 修改3: 移除了有风险的 "首次命中即中断" 的 break 语句
            }

            float sh = 0.0f;
            if (f == h_orig) { // 如果最大视线高度没有超过起始点高度，则天空可见
                sh = 1.0f;
            }
            // --------------------

            for (int k = (int)annulino[i] + 1; k < (int)annulino[i + 1] + 1; k++) {
                float weight = annulus_weight((float)k, aziinterval[i]);
                weight *= sh;
                svf += weight;
            }
            idx++;
        }
    }
    svf_Latt[index4] = svf;
}
